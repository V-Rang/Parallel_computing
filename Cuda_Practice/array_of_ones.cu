#include<iostream>
#include<hip/hip_runtime.h>


using namespace std;

__global__ void init(int *test, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < n) test[tid] = 1;
}


int main()
{
	int n = 16;
	int *h_x = new int[n];
	for(int i=0;i<n;i++) h_x[i] = 0;

	int *d_x;

	hipMalloc(&d_x,n*sizeof(int));
	hipMemcpy(d_x,h_x,n*sizeof(int),hipMemcpyHostToDevice);
	for(int i=0;i<n;i++) cout << h_x[i] <<  " ";
	cout << endl;


	init<<<1,32>>>(d_x,n);

	hipMemcpy(h_x,d_x,n*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++) cout << h_x[i] <<  " ";
	cout << endl;
	return 0;
	
}
