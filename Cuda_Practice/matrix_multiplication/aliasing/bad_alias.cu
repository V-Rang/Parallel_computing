/*
(transposing B and then traversing the elements of its rows, i.e. each thread accesses a unique row of Bt, each of which are displaced far 
from each other in memory. This results in bad memory usage as data for threads cannot be found with single call to DRAM that will load the 
values needed by the threads in the cache.)
*/

#include<iostream>
#include<hip/hip_runtime.h>

#include<vector>
#include<algorithm>
#include<assert.h>

using namespace std;

__global__ void matrix_mult(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    c[row*n + col] = 0;

    for(int i=0;i<n;i++) c[row*n+col] += a[row*n + i]*b[col*n + i];
}

void verify_result(vector<int>&a, vector<int>&b, vector<int>&c, int n)
{
    for(int i=0;i<n;i++) //for each row
    {
        for(int j=0;j<n;j++) //for each col
        {
            int temp = 0;
            for(int k=0;k<n;k++)
            {
                temp += a[i*n + k]*b[k*n + j];
            }
            assert(temp == c[i*n+j]);
        }
    }
}

void transposer(vector<int>&a, vector<int>&a_t,int n)
{
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            a_t[j*n+i] = a[i*n+j];
        }
    }
}

int main()
{
    int n = 1 << 10;
    int num_threads = 32;
    // int n = 4;
    // int num_threads = 2;
    int num_blocks = n/num_threads;
    size_t size = n * n * sizeof(int);

    dim3 threads(num_threads,num_threads);
    dim3 blocks(num_blocks,num_blocks);

    vector<int>h_a(n*n), h_b(n*n), h_c(n*n);

    vector<int>h_b_t(n*n);

    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);  

    generate(h_a.begin(), h_a.end(), [](){return rand()%100;});
    generate(h_b.begin(), h_b.end(), [](){return rand()%100;});

    transposer(h_b,h_b_t,n);


    hipMemcpy(d_a,h_a.data(),size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b_t.data(),size,hipMemcpyHostToDevice);
    
    matrix_mult<<<blocks,threads>>>(d_a,d_b,d_c,n);

    hipMemcpy(h_c.data(),d_c,size,hipMemcpyDeviceToHost);

    verify_result(h_a,h_b,h_c,n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    

    printf("Successful\n");

    return 0;
}
