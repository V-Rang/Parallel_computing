/*
good aliasing:
Good: At X B (transposing A and then traversing the elements of its columns, i.e. each thread accesses a unique column of At, each of which 
are next to each other in memory. This results in good memory usage as data for threads can be found with single call to DRAM that will load the 
values needed by the threads in the cache.)
Time: using NsightCompute: 87.56 ms
*/

#include<iostream>
#include<hip/hip_runtime.h>

#include<vector>
#include<algorithm>
#include<assert.h>

using namespace std;

__global__ void matrix_mult(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    c[row*n + col] = 0;

    for(int i=0;i<n;i++) c[row*n+col] += a[row*n + i]*b[i*n + col];

}

void verify_result(vector<int>&a, vector<int>&b, vector<int>&c, int n)
{
    for(int i=0;i<n;i++) //for each row
    {
        for(int j=0;j<n;j++) //for each col
        {
            int temp = 0;
            for(int k=0;k<n;k++)
            {
                temp += a[i*n + k]*b[k*n + j];
            }
            assert(temp == c[i*n+j]);
        }
    }
}

void transposer(vector<int>&a, vector<int>&a_t,int n)
{
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            a_t[j*n+i] = a[i*n+j];
        }
    }
}

int main()
{
    int n = 1 << 10;
    int num_threads = 32;
    // int n = 4;
    // int num_threads = 2;
    int num_blocks = n/num_threads;
    size_t size = n * n * sizeof(int);

    dim3 threads(num_threads,num_threads);
    dim3 blocks(num_blocks,num_blocks);

    vector<int>h_a(n*n), h_b(n*n), h_c(n*n),h_a_t(n*n);

    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);  

    generate(h_a.begin(), h_a.end(), [](){return rand()%100;});
    generate(h_b.begin(), h_b.end(), [](){return rand()%100;});

    transposer(h_a,h_a_t,n);

    

    hipMemcpy(d_a,h_a.data(),size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b.data(),size,hipMemcpyHostToDevice);
    

    matrix_mult<<<blocks,threads>>>(d_a,d_b,d_c,n);

    hipMemcpy(h_c.data(),d_c,size,hipMemcpyDeviceToHost);


    verify_result(h_a,h_b,h_c,n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    

    printf("Successful\n");

    return 0;
}
