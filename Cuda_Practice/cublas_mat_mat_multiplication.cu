
#include <hip/hip_runtime.h>
#include<iostream>
#include<hipblas.h>
#include<assert.h>
#include<hiprand/hiprand.h>

using namespace std;

void verify_results(float *a, float *b, float *c, int n)
{
    float temp;
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            temp = 0;
            for(int k = 0;k<n;k++)
            {
                temp += a[k*n + j] * b[i*n + k];
            }
            assert(fabs(temp - c[i*n+j]) < 1e-3);
        }
    }
}

// void verify_solution(float *a, float *b, float *c, int n)
// {
//     float temp;
//     float epsilon = 0.001;
//     for(int i=0;i<n;i++)
//     {
//         for(int j=0;j<n;j++)
//         {
//             temp = 0;
//             for(int k=0;k<n;k++)
//             {
//                 temp += a[k*n + i]*b[j*n + k];
//             }
//             assert(fabs(c[j*n + i] - temp) < epsilon);
//         }
//     }
// }

int main()
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // int n = 2;
    int n = 1 << 10;
    size_t size = n*n*sizeof(float);

    h_a = new float[n*n];
    h_b = new float[n*n];
    h_c = new float[n*n];

    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng,(unsigned long long)clock());

    hiprandGenerateUniform(prng,d_a,size);
    hiprandGenerateUniform(prng,d_b,size);
     
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1;
    float beta = 0;

    cublasSgemm('N','N',n,n,n,alpha,d_a,n,d_b,n,beta,d_c,n);

    hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);
    hipMemcpy(h_a,d_a,size,hipMemcpyDeviceToHost);
    hipMemcpy(h_b,d_b,size,hipMemcpyDeviceToHost);

    //C++ verification code
    verify_results(h_a,h_b,h_c,n);
    printf("Successful\n");


    return 0;
}
