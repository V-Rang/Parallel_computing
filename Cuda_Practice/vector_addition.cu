#include<hip/hip_runtime.h>

#include<iostream>

using namespace std;

__global__ void vectorAdd(float *x, float *y, float *z, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < n) z[tid] = x[tid] + y[tid];
}

int main()
{
	float *a, *b, *c;
	float *d_a,*d_b,*d_c;

	int n = 5;

	a = new float[5]{1,2,3,4,5};
	b = new float[5]{11,12,13,14,15};

	c = new float[5];

	int size = n * sizeof(float);

	hipMalloc(&d_a,size);
	hipMalloc(&d_b,size);
	hipMalloc(&d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);
	
	vectorAdd<<<1,1024>>>(d_a,d_b,d_c,n);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++) std::cout << c[i] << " ";
	std::cout << endl;
	


	return 0;
}
